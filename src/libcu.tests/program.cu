#include <cuda_runtimecu.h>
#include <sentinel.h>
#include <stdiocu.h>

hipError_t fcntl_test1();
#define test fcntl_test1

int main()
{
	sentinelServerInitialize();

	// Choose which GPU to run on, change this on a multi-GPU system.
	hipError_t cudaStatus = hipSetDevice(gpuGetMaxGflopsDevice());
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Launch test
	cudaStatus = test();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

Error:
	sentinelServerShutdown();

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}
