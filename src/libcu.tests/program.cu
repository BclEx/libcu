#include <cuda_runtimecu.h>
#include <sentinel.h>
#include <stdlibcu.h>
#include <stdiocu.h>

hipError_t crtdefs_test1();
hipError_t ctype_test1();
hipError_t dirent_test1();
hipError_t errno_test1();
hipError_t falloc_lauched_cuda_kernel();
hipError_t falloc_alloc_with_getchunk();
hipError_t falloc_alloc_with_getchunks();
hipError_t falloc_alloc_with_context();
hipError_t fcntl_test1();
hipError_t fsystem_test1();
hipError_t grp_test1();
hipError_t pwd_test1();
hipError_t regex_test1();
hipError_t sentinel_test1();
hipError_t setjmp_test1();
hipError_t stdarg_parse();
hipError_t stdarg_call();
hipError_t stddef_test1();
hipError_t stdio_test1();
hipError_t stdio_64bit();
hipError_t stdio_ganging();
hipError_t stdio_scanf();
hipError_t stdlib_test1();
hipError_t stdlib_strtol();
hipError_t stdlib_strtoq();
hipError_t string_test1();
hipError_t time_test1();
hipError_t unistd_test1();

#if _HASPAUSE
#define mainPause(fmt) { printf(fmt"\n"); char c; scanf("%c", &c); }
#else
#define mainPause(fmt) { printf(fmt"\n"); }
#endif

int main(int argc, char ** argv)
{
	int testId = argv[1] ? atoi(argv[1]) : 25; //0; //18; //25;

	// Choose which GPU to run on, change this on a multi-GPU system.
	hipError_t cudaStatus = hipSetDevice(gpuGetMaxGflopsDevice());
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
		goto Error;
	}
	cudaErrorCheck(hipDeviceSetLimit(hipLimitStackSize, 1024*5));
	sentinelServerInitialize();
	sentinelRegisterFileUtils();

	// Launch test
	switch (testId)
	{
	case 0: mainPause("Press any key to continue."); break;
	case 1: cudaStatus = crtdefs_test1(); break;
	case 2: cudaStatus = ctype_test1(); break;
	case 3: cudaStatus = dirent_test1(); break;
	case 4: cudaStatus = errno_test1(); break;
	case 5: cudaStatus = falloc_lauched_cuda_kernel(); break;
	case 6: cudaStatus = falloc_alloc_with_getchunk(); break; // memory access
	case 7: cudaStatus = falloc_alloc_with_getchunks(); break;
	case 8: cudaStatus = falloc_alloc_with_context(); break; // memory access
	case 9: cudaStatus = fcntl_test1(); break;
	case 10: cudaStatus = grp_test1(); break;
	case 11: cudaStatus = pwd_test1(); break;
	case 12: cudaStatus = regex_test1(); break;
	case 13: cudaStatus = sentinel_test1(); break;
	case 14: cudaStatus = setjmp_test1(); break;
	case 15: cudaStatus = stdarg_parse(); break;
	case 16: cudaStatus = stdarg_call(); break;
	case 17: cudaStatus = stddef_test1(); break;
	case 18: cudaStatus = stdio_test1(); break; // assert
	case 19: cudaStatus = stdio_64bit(); break;
	case 20: cudaStatus = stdio_ganging(); break;
	case 21: cudaStatus = stdio_scanf(); break;
	case 22: cudaStatus = stdlib_test1(); break; // assert
	case 23: cudaStatus = stdlib_strtol(); break;
	case 24: cudaStatus = stdlib_strtoq(); break;
	case 25: cudaStatus = string_test1(); break;
	case 26: cudaStatus = time_test1(); break;
	case 27: cudaStatus = unistd_test1(); break; // missing device, throws on fast run
		// default
	default: cudaStatus = crtdefs_test1(); break;
	}
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// finish
	mainPause("SUCCESS");

Error:
	sentinelServerShutdown();

	// close
	if (cudaStatus != hipSuccess) {
		// finish
		mainPause("ERROR");
		return 1;
	}

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!\n");
		return 1;
	}

	return 0;
}
