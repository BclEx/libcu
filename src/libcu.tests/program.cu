#include <cuda_runtimecu.h>
#include <sentinel.h>
#include <stdiocu.h>

hipError_t crtdefs_test1();
hipError_t ctype_test1();
hipError_t dirent_test1();
hipError_t errno_test1();
hipError_t falloc_lauched_cuda_kernel();
hipError_t falloc_alloc_with_getchunk();
hipError_t falloc_alloc_with_getchunks();
hipError_t falloc_alloc_with_context();
hipError_t fcntl_test1(); // fails
hipError_t fsystem_test1();
hipError_t grp_test1();
hipError_t pwd_test1();
hipError_t regex_test1();
hipError_t sentinel_test1();
hipError_t setjmp_test1();
hipError_t stdarg_parse();
hipError_t stdarg_call();
hipError_t stddef_test1();
hipError_t stdio_test1(); // fails
hipError_t stdio_64bit();
hipError_t stdio_ganging();
hipError_t stdio_scanf();
hipError_t stdlib_test1(); // fails
hipError_t stdlib_strtol();
hipError_t stdlib_strtoq();
hipError_t string_test1();
hipError_t time_test1();
hipError_t unistd_test1();

#define test stdlib_test1

int main()
{
	sentinelServerInitialize();

	// Choose which GPU to run on, change this on a multi-GPU system.
	hipError_t cudaStatus = hipSetDevice(gpuGetMaxGflopsDevice());
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
		goto Error;
	}
	cudaErrorCheck(hipDeviceSetLimit(hipLimitStackSize, 1024*5));

	// Launch test
	cudaStatus = test();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "test failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "test launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

Error:
	sentinelServerShutdown();

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!\n");
		return 1;
	}

	// finish
	printf("\nPress any key to continue.\n");
	scanf("%c");

	return 0;
}
