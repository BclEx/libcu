#include <cuda_runtimecu.h>
#include <sentinel.h>
#include <stdio.h>

int all_start(bool sentinel)
{
	if (sentinel)
		sentinelServerInitialize();

	// Choose which GPU to run on, change this on a multi-GPU system.
	hipError_t cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return 1;
	}
	return 0;
}

int all_after()
{
	// Check for any errors launching the kernel
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		return 1;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		return 1;
	}
	return 0;
}

int all_end(bool sentinel)
{
	if (sentinel)
		sentinelServerShutdown();

	// hipDeviceReset must be called before exiting in order for profiling and tracing tools such as Nsight and Visual Profiler to show complete traces.
	hipError_t cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}
	return 0;
}