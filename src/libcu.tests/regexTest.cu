#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdiocu.h>
#include <regexcu.h>
#include <assert.h>

static __global__ void g_regex_test1()
{
	printf("regex_test1\n");
}
hipError_t regex_test1() { g_regex_test1<<<1, 1>>>(); return hipDeviceSynchronize(); }
