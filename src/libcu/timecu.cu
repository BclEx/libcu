#include "hip/hip_runtime.h"
#include <timecu.h>

__BEGIN_DECLS;
#if defined(__CUDA_ARCH__) || defined(LIBCUFORCE)

// time
__device__ time_t time_(time_t *timer)
{
	clock_t start = clock();
	time_t epoch = 0;
	return epoch;
}

// gettimeofday
__device__ int gettimeofday_(struct timeval *tp, void *tz)
{
	time_t seconds = time(nullptr);
	tp->tv_usec = 0;
	tp->tv_sec = seconds;
	return 0;
	//if (tz)
	//	_abort();
	//tp->tv_usec = 0;
	//return (_time(&tp->tv_sec) == (time_t)-1 ? -1 : 0);
}

#else
#ifdef _MSC_VER
#include <sys/timeb.h>
int gettimeofday(struct timeval *tv, void *unused)
{
	struct _timeb tb;
	_ftime(&tb);
	tv->tv_sec = tb.time;
	tv->tv_usec = tb.millitm * 1000;
	return 0;
}
#endif
#endif
__END_DECLS;