#include "hip/hip_runtime.h"
#include <timecu.h>

__BEGIN_DECLS;
#if defined(__CUDA_ARCH__) || defined(LIBCUFORCE)

// time
__device__ time_t time_(time_t *timer)
{
	clock_t start = clock();
	time_t epoch = 0;
	return epoch;
}

// gettimeofday
__device__ int gettimeofday_(struct timeval *tp, void *tz)
{
	time_t seconds = time(nullptr);
	tp->tv_usec = 0;
	tp->tv_sec = seconds;
	return 0;
	//if (tz)
	//	_abort();
	//tp->tv_usec = 0;
	//return (_time(&tp->tv_sec) == (time_t)-1 ? -1 : 0);
}

#endif
__END_DECLS;