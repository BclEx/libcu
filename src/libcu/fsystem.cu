#include "hip/hip_runtime.h"
#include <stdiocu.h>
#include <stdlibcu.h>
#include <stddefcu.h>
#include <assert.h>
#include <ext/hash.h>
#include <errnocu.h>
#include "fsystem.h"

__BEGIN_DECLS;

// FILES
#pragma region FILES

typedef struct __align__(8)
{
	file_t *file;			// reference
	unsigned short id;		// ID of author
	unsigned short threadid;// thread ID of author
} fileRef;

__device__ fileRef __iob_fileRefs[CORE_MAXFILESTREAM]; // Start of circular buffer (set up by host)
volatile __device__ fileRef *__iob_freeFilePtr = __iob_fileRefs; // Current atomically-incremented non-wrapped offset
volatile __device__ fileRef *__iob_retnFilePtr = __iob_fileRefs; // Current atomically-incremented non-wrapped offset
__constant__ file_t __iob_files[CORE_MAXFILESTREAM];

static __device__ __forceinline void writeFileRef(fileRef *ref, file_t *f)
{
	ref->file = f;
	ref->id = gridDim.x*blockIdx.y + blockIdx.x;
	ref->threadid = blockDim.x*blockDim.y*threadIdx.z + blockDim.x*threadIdx.y + threadIdx.x;
}

static __device__ int fileGet(file_t **file)
{
	// advance circular buffer
	size_t offset = (atomicAdd((uintptr_t *)&__iob_freeFilePtr, sizeof(fileRef)) - (size_t)&__iob_fileRefs);
	offset %= (sizeof(fileRef)*CORE_MAXFILESTREAM);
	int offsetId = offset / sizeof(fileRef);
	fileRef *ref = (fileRef *)((char *)&__iob_fileRefs + offset);
	file_t *f = ref->file;
	if (!f) {
		f = &__iob_files[offsetId];
		writeFileRef(ref, f);
	}
	*file = f;
	return GETFD(offsetId);
}

static __device__ void fileFree(int fd)
{
	//if (!f) return;
	file_t *f = GETFILE(fd);
	// advance circular buffer
	size_t offset = atomicAdd((uintptr_t *)&__iob_retnFilePtr, sizeof(fileRef)) - (size_t)&__iob_fileRefs;
	offset %= (sizeof(fileRef)*CORE_MAXFILESTREAM);
	fileRef *ref = (fileRef *)((char *)&__iob_fileRefs + offset);
	writeFileRef(ref, f);
}

#pragma endregion

__device__ char __cwd[MAX_PATH] = ":\\";
__device__ dirEnt_t __iob_root = { { 0, 0, 0, 1, ":\\" }, nullptr, nullptr };
__device__ hash_t __iob_dir = HASHINIT;

__device__ void expandPath(const char *path, char *newPath)
{
	register unsigned char *d = (unsigned char *)newPath;
	register unsigned char *s;
	// add cwd
	if (path[0] != ':') {
		s = (unsigned char *)__cwd;
		while (*s) { *d++ = *s++; }
	}
	// add path
	s = (unsigned char *)path;
	int i = 0;
	while (*s) {
		int c = *s;
		if (c == '/') c = '\\'; // switch from unix path
		if (c == '\\') {
			// directory reached
			if (i == 1 && s[-1] == '.') d -= 2; // self directory
			else if (i == 2 && s[-1] == '.' && s[-2] == '.') while (*d >= *newPath && *d != '\\') *d--; // parent directory
			i = 0;
		}
		// advance
		*d++ = c; s++; i++;
	}
	// remove trailing '\'
	d[*d == '\\' ? 0 : 1] = 0;
}

static __device__ void freeEnt(dirEnt_t *ent)
{
	if (ent->dir.d_type == 1) {
		dirEnt_t *p = ent->u.list;
		while (p) {
			dirEnt_t *next = p->next;
			freeEnt(p);
			p = next;
		}
	} else if (ent->dir.d_type == 2)
		memfileClose(ent->u.file);
	if (ent != &__iob_root) free(ent);
	else __iob_root.u.list = nullptr;
}

static __device__ dirEnt_t *findDir(const char *path, const char **file)
{
	char *file2 = strrchr((char *)path, '\\');
	if (!file2) {
		_set_errno(EINVAL);
		return nullptr;
	}
	*file2 = 0;
	dirEnt_t *ent = !strcmp(path, ":")
		? &__iob_root 
		: (dirEnt_t *)hashFind(&__iob_dir, path);
	*file2 = '\\';
	*file = file2 + 1;
	return ent;
}

__device__ int fsystemRename(const char *old, const char *new_)
{
	char newPath[MAX_PATH]; expandPath(old, newPath);
	dirEnt_t *ent = (dirEnt_t *)hashFind(&__iob_dir, old);
	if (!ent) {
		_set_errno(ENOENT);
		return -1;
	}
	return 0;
}

__device__ int fsystemUnlink(const char *path)
{
	char newPath[MAX_PATH]; expandPath(path, newPath);
	dirEnt_t *ent = (dirEnt_t *)hashFind(&__iob_dir, newPath);
	if (!ent) {
		_set_errno(ENOENT);
		return -1;
	}
	const char *name;
	dirEnt_t *parentEnt = findDir(newPath, &name);
	if (!parentEnt) {
		_set_errno(ENOENT);
		return -1;
	}

	//// directory not empty
	//if (ent->dir.d_type == 1 && ent->list) {
	//	_set_errno(ENOENT);
	//	return -1;
	//}

	// remove from directory
	dirEnt_t *list = parentEnt->u.list;
	if (list == ent)
		parentEnt->u.list = ent->next;
	else if (list) {
		dirEnt_t *p = list;
		while (p->next && p->next != ent)
			p = p->next;
		if (p->next == ent)
			p->next = ent->next;
	}

	// free entity
	freeEnt(ent);
	return 0;
}

__device__ dirEnt_t *fsystemMkdir(const char *__restrict path, int mode, int *r)
{
	char newPath[MAX_PATH]; expandPath(path, newPath);
	dirEnt_t *dirEnt = (dirEnt_t *)hashFind(&__iob_dir, newPath);
	if (dirEnt) {
		*r = 1;
		return dirEnt;
	}
	const char *name;
	dirEnt_t *parentEnt = findDir(newPath, &name);
	if (!parentEnt) {
		_set_errno(ENOENT);
		*r = -1;
		return nullptr;
	}
	// create directory
	dirEnt = (dirEnt_t *)malloc(sizeof(dirEnt_t));
	if (hashInsert(&__iob_dir, newPath, dirEnt))
		panic("removed directory");
	dirEnt->dir.d_type = 1;
	strcpy(dirEnt->dir.d_name, name);
	// add to directory
	dirEnt->next = parentEnt->u.list; parentEnt->u.list = dirEnt;
	*r = 0;
	return dirEnt;
}

__device__ dirEnt_t *fsystemOpen(const char *__restrict path, int mode, int *fd)
{
	char newPath[MAX_PATH]; expandPath(path, newPath);
	dirEnt_t *fileEnt = (dirEnt_t *)hashFind(&__iob_dir, newPath);
	if (fileEnt) {
		file_t *f; *fd = fileGet(&f);
		f->base = (char *)fileEnt;
		return fileEnt;
	}
	if ((mode & 0xF) == O_RDONLY) {
		_set_errno(EINVAL); // So illegal mode.
		*fd = -1;
		return nullptr;
	}
	const char *name;
	dirEnt_t *parentEnt = findDir(newPath, &name);
	if (!parentEnt) {
		_set_errno(ENOENT);
		*fd = -1;
		return nullptr;
	}
	// create file
	fileEnt = (dirEnt_t *)malloc(_ROUND64(sizeof(dirEnt_t)) + __sizeofMemfile_t);
	if (hashInsert(&__iob_dir, newPath, fileEnt))
		panic("removed file");
	fileEnt->dir.d_type = 2;
	strcpy(fileEnt->dir.d_name, name);
	fileEnt->u.file = (memfile_t *)((char *)fileEnt + _ROUND64(sizeof(dirEnt_t)));
	memfileOpen(fileEnt->u.file);
	// add to directory
	fileEnt->next = parentEnt->u.list; parentEnt->u.list = fileEnt;
	// set to file
	file_t *f; *fd = fileGet(&f);
	f->base = (char *)fileEnt;
	return fileEnt;
}

__device__ void fsystemReset()
{
	freeEnt(&__iob_root);
}

__END_DECLS;