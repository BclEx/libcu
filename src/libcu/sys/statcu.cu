#include <sys/statcu.h>
#include <sentinel-fcntlmsg.h>
#include "../fsystem.h"

/* Get file attributes for FILE and put them in BUF.  */
__device__ int stat_(const char *__restrict file, struct stat *__restrict buf, bool lstat)
{
	if (!ISDEVICEPATH(file)) {
		fcntl_stat msg(file, buf, lstat); return msg.RC;
	}
	panic("Not Implemented");
	return 0;
}

/* Get file attributes for the file, device, pipe, or socket that file descriptor FD is open on and put them in BUF.  */
__device__ int fstat_device(int fd, struct stat *buf)
{
	panic("Not Implemented");
	return 0;
}

#ifdef __USE_LARGEFILE64
/* Get file attributes for FILE and put them in BUF.  */
__device__ int stat64_(const char *__restrict file, struct stat64 *__restrict buf, bool lstat)
{
	if (!ISDEVICEPATH(file)) {
		fcntl_stat64 msg(file, buf, lstat); return msg.RC;
	}
	panic("Not Implemented");
	return 0;
}

/* Get file attributes for the file, device, pipe, or socket that file descriptor FD is open on and put them in BUF.  */
__device__ int fstat64_device(int fd, struct stat64 *buf)
{
	panic("Not Implemented");
	return 0;
}
#endif

/* Set file access permissions for FILE to MODE. If FILE is a symbolic link, this affects its target instead.  */
__device__ int chmod_(const char *file, mode_t mode)
{
	panic("Not Implemented");
	return 0;
}

/* Set the file creation mask of the current process to MASK, and return the old creation mask.  */
__device__ mode_t umask_(mode_t mask)
{
	panic("Not Implemented");
	return 0;
}

/* Create a new directory named PATH, with permission bits MODE.  */
__device__ int mkdir_(const char *path, mode_t mode)
{
	if (!ISDEVICEPATH(path)) {
		fcntl_mkdir msg(path, mode); return msg.RC;
	}
	int r; fsystemMkdir(path, mode, &r); return r;
}

/* Create a new FIFO named PATH, with permission bits MODE.  */
__device__ int mkfifo_(const char *path, mode_t mode)
{
	panic("Not Implemented");
	return 0;
}
