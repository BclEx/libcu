#include "hip/hip_runtime.h"
#include <cuda_runtimecu.h>
#include <stdiocu.h>
#include <stringcu.h>
#include <assert.h>

static __global__ void g_general_speed()
{
	char *test = 
		"The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog."
		"The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog."
		"The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog."
		"The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog."
		"The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog."
		"The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog."
		"The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog."
		"The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog."
		"The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog."
		"The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog."
		"The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog."
		"The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog.";
	for (int i = 0; i < 1000; i++) {
		int testLength = strnlen(nullptr, 3000);
		assert(testLength == 0);
	}
	for (int i = 0; i < 1000; i++) {
		int testLength = strnlen(test, 3000);
		assert(testLength == 2196);
		//printf("%d\n", testLength);
	}
}

int main()
{
	float milliseconds = 0;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Choose which GPU to run on, change this on a multi-GPU system.
	hipError_t cudaStatus = hipSetDevice(gpuGetMaxGflopsDevice());
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
		goto Error;
	}

	// Launch test
	hipEventRecord(start);
	for (int i = 0; i < 1; i++)
		g_general_speed<<<1, 32>>>();
	hipEventRecord(stop);
	//
	cudaStatus = hipDeviceSynchronize(); 
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "test failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "test launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("Effective: %fn", milliseconds/1e6);

Error:
	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!\n");
		return 1;
	}

	// finish
	printf("\nPress any key to continue.\n");
	scanf("%c");

	return 0;
}
