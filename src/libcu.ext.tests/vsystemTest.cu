#include "hip/hip_runtime.h"
#include <stdiocu.h>
#include <crtdefscu.h>
#include <ext\global.h>
#include <assert.h>

static __global__ void g_vsystem_test1()
{
	printf("vsystem_test1\n");
}
hipError_t vsystem_test1() { g_vsystem_test1<<<1, 1>>>(); return hipDeviceSynchronize(); }