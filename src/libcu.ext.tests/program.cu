#include <cuda_runtimecu.h>
#include <sentinel.h>
#include <stdlibcu.h>
#include <stdiocu.h>
#include <ext\global.h>

hipError_t alloc_test1();
hipError_t bitvec_test1();
hipError_t convert_test1();
hipError_t global_test1();
hipError_t main_test1();
hipError_t math_test1();
hipError_t mutex_test1();
hipError_t notify_test1();
hipError_t pcache_test1();
hipError_t pcache1_test1();
hipError_t printf_test1();
hipError_t random_test1();
hipError_t status_test1();
hipError_t utf_test1();
hipError_t util_test1();
hipError_t vsystem_test1();

#if _HASPAUSE
#define mainPause(fmt) { printf(fmt"\n"); char c; scanf("%c", &c); }
#else
#define mainPause(fmt) { printf(fmt"\n"); }
#endif

int main(int argc, char **argv)
{
	int testId = argv[1] ? atoi(argv[1]) : 0;

	// Choose which GPU to run on, change this on a multi-GPU system.
	hipError_t cudaStatus = hipSetDevice(gpuGetMaxGflopsDevice());
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
		goto Error;
	}
	cudaErrorCheck(hipDeviceSetLimit(hipLimitStackSize, 1024*5));
	sentinelServerInitialize();
	runtimeInitialize();

	// Launch test
	switch (testId)
	{
	case 0: mainPause("Press any key to continue."); break;
	case 1: cudaStatus = alloc_test1(); break;
	case 2: cudaStatus = bitvec_test1(); break;
	case 3: cudaStatus = convert_test1(); break;
	case 4: cudaStatus = global_test1(); break;
	case 5: cudaStatus = math_test1(); break;
	case 6: cudaStatus = main_test1(); break;
	case 7: cudaStatus = mutex_test1(); break;
	case 8: cudaStatus = notify_test1(); break;
	case 9: cudaStatus = pcache_test1(); break;
	case 10: cudaStatus = pcache1_test1(); break;
	case 11: cudaStatus = printf_test1(); break;
	case 12: cudaStatus = random_test1(); break;
	case 13: cudaStatus = status_test1(); break;
	case 14: cudaStatus = utf_test1(); break;
	case 15: cudaStatus = util_test1(); break;
	case 16: cudaStatus = vsystem_test1(); break;
		// default
	default: cudaStatus = bitvec_test1(); break;
	}
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// finish
	mainPause("SUCCESS");

Error:
	runtimeShutdown();
	sentinelServerShutdown();

	// close
	if (cudaStatus != hipSuccess) {
		// finish
		mainPause("ERROR");
		return 1;
	}

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!\n");
		return 1;
	}

	return 0;
}
