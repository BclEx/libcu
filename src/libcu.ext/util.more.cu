﻿/*
** Return the declared type of a column.  Or return zDflt if the column 
** has no declared type.
**
** The column type is an extra string stored after the zero-terminator on
** the column name if and only if the COLFLAG_HASTYPE flag is set.
*/
char *sqlite3ColumnType(Column *pCol, char *zDflt){
  if( (pCol->colFlags & COLFLAG_HASTYPE)==0 ) return zDflt;
  return pCol->zName + strlen(pCol->zName) + 1;
}