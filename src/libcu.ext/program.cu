#include "hip/hip_runtime.h"
#include <cuda_runtimecu.h>
#include <sentinel.h>
#include <stdlibcu.h>
#include <stdiocu.h>
#include <ext\global.h>

static __global__ void g_test1()
{
	printf("test1\n");

	void *a = alloc32(10);
	mfree(a);
}
hipError_t test1() { g_test1<<<1, 1>>>(); return hipDeviceSynchronize(); }

#if _HASPAUSE
#define mainPause(fmt) { printf(fmt"\n"); char c; scanf("%c", &c); }
#else
#define mainPause(fmt) { printf(fmt"\n"); }
#endif

int main(int argc, char ** argv)
{
	int testId = argv[1] ? atoi(argv[1]) : 1;

	// Choose which GPU to run on, change this on a multi-GPU system.
	hipError_t cudaStatus = hipSetDevice(gpuGetMaxGflopsDevice());
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
		goto Error;
	}
	cudaErrorCheck(hipDeviceSetLimit(hipLimitStackSize, 1024*5));
	sentinelServerInitialize();

	// Launch test
	switch (testId)
	{
	case 0: mainPause("Press any key to continue."); break;
	case 1: cudaStatus = test1(); break;
	default: break;
	}
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// finish
	mainPause("SUCCESS");

Error:
	sentinelServerShutdown();

	// close
	if (cudaStatus != hipSuccess) {
		// finish
		mainPause("ERROR");
		return 1;
	}

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!\n");
		return 1;
	}

	return 0;
}
