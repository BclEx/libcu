#include "hip/hip_runtime.h"
#include <stdiocu.h>
#include <errnocu.h>

#define CAT_BUF_SIZE 4096

char colon[2] = { ':', ' ' };
char nl = '\n';

__device__ void dumpfile(FILE *f)
{
	int nred;
	char readbuf[CAT_BUF_SIZE];
	while ((nred = fread(readbuf, 1, CAT_BUF_SIZE, f)) > 0)
		fwrite(readbuf, nred, 1, stdout);
}

__device__ __managed__ int m_dcat_rc;
__global__ void g_dcat(char *str)
{
	FILE *f = fopen(str, "r");
	if (!f)
		m_dcat_rc = errno;
	else
	{
		dumpfile(f);
		fclose(f);
		m_dcat_rc = 0;
	}
}

int dcat(char *str)
{
	g_dcat<<<1,1>>>(str);
	return m_dcat_rc;
}

/*
int dcat(char *str)
{
	g_dcat<<<1,1>>>(str);
	int rc;
	hipMemcpyFromSymbol(&rc, HIP_SYMBOL("d_rc"), sizeof(rc), 0, hipMemcpyDeviceToHost);
	return rc;
}
*/