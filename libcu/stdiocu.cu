#include <stdiocu.h>
#include <cuda_runtimecu.h>
#include <sentinel-stdiomsg.h>

//__device__ int _close(int a) { io_close msg(a); return msg.RC; }

__BEGIN_DECLS;

__constant__ FILE *__iob_file[3] = { (FILE *)1, (FILE *)2, (FILE *)3 };
//__device__ FILE *stdin;
//__device__ FILE *stdout;
//__device__ FILE *stderr;

__END_DECLS;

/* Write formatted output to S from argument list ARG. */
__device__ int vfprintf(FILE *__restrict s, const char *__restrict format, va_list arg) { return -1; }
//__device__ int fprintf(FILE *f, const char *v, bool wait) { stdio_fprintf msg(wait, f, v); _free((void *)v); return msg.RC; }

/* Write formatted output to stdout from argument list ARG. */
__device__ int vprintf(const char *__restrict format, va_list arg) { return -1; }
/* Write formatted output to S from argument list ARG.  */
__device__ int vsprintf(char *__restrict s, const char *__restrict format, va_list arg) { return -1; }

__device__ int vsnprintf(char *__restrict s, size_t maxlen, const char *__restrict format, va_list arg) { return -1; }

/* Read formatted input from S into argument list ARG.  */
__device__ int vfscanf(FILE *__restrict s, const char *__restrict format, va_list arg) { return -1; }
/* Read formatted input from stdin into argument list ARG. */
__device__ int vscanf(const char *__restrict format, va_list arg) { return -1; }
/* Read formatted input from S into argument list ARG.  */
__device__ int vsscanf(const char *__restrict s, const char *__restrict format, va_list arg) { return -1; }

/* Read a character from STREAM.  */
//__device__ int getc(FILE *stream) { return -1; }

/* Read a character from stdin.  */
__device__ int getchar(void) { return -1; }

/* Write a character to STREAM.  */
//__device__ int putc(int c, FILE *stream) { return -1; }

/* Write a character to stdout.  */
__device__ int putchar(int c) { return -1; }

/* Write a string, followed by a newline, to stdout.  */
__device__ int puts(const char *s) { printf("%s\n", s); return -1; }

/* Push a character back onto the input buffer of STREAM.  */
__device__ int ungetc(int c, FILE *stream) { return -1; }

/* Print a message describing the meaning of the value of errno.  */
//__device__ void perror(const char *s) { }
